#include "hip/hip_runtime.h"
extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_skein.h"
#include "sph/sph_keccak.h"
#include "sph/sph_cubehash.h"
#include "lyra2/Lyra2.h"
#include "nightcap/nightcap.h"
}

#include <miner.h>
#include <cuda_helper.h>

static uint64_t *d_hash[MAX_GPUS];
static uint64_t* d_matrix[MAX_GPUS];

typedef union _NCMixNode {
	uint32_t values[16];
	uint4 nodes4[4];
} NCMixNode;

typedef union _NCLightNode {
	uint32_t values[8];
	uint4 nodes4[2];
} NCLightNode;

// statics
__constant__ uint32_t nc_d_dag_size;
__constant__ NCMixNode* nc_d_dag;
__constant__ uint32_t nc_d_light_size;
__constant__ NCLightNode* nc_d_light;
__constant__ uint32_t nc_d_height;


// State per GPU
typedef struct _NCGPUState
{
	NCLightNode* cache_nodes;
	NCMixNode* dag_nodes;

	uint64_t dag_size;
	uint64_t cache_size;

	uint64_t num_dag_nodes;
	uint64_t num_cache_nodes;

	uint32_t epoch;
	uint32_t height;
} NCGPUState;


// State per thread (to keep track of when we need to reset cuda vars)
typedef struct _NCThreadState
{
	uint32_t epoch;
} NCThreadState;


static NCGPUState nc_gpu_state[MAX_GPUS];
static NCThreadState nc_thread_state[MAX_GPUS];

extern void blake256_cpu_init(int thr_id, uint32_t threads);
extern void blake256_cpu_setBlock_80(uint32_t *pdata);

extern void blakeKeccak256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash, int order);

extern void skein256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);
extern void skein256_cpu_init(int thr_id, uint32_t threads);
extern void cubehash256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_hash, int order);

extern void lyra2v2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash, int order);
extern void lyra2v2_cpu_init(int thr_id, uint32_t threads, uint64_t* d_matrix);

extern void bmw256_setTarget(const void *ptarget);
extern void bmw256_cpu_init(int thr_id, uint32_t threads);
extern void bmw256_cpu_free(int thr_id);
extern void bmw256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *resultnonces);
extern void bmw256_cpu_hash_32_to_hash(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *g_hash, int order);

__device__ __inline__ uint32_t fnv(const uint32_t v1, const uint32_t v2) {
 return (((v1 * NIGHTCAP_FNV_PRIME) ^ v2) % (0xFFFFFFFF));
}

__device__ __inline__ uint4 fnv4(uint4 a, uint4 b)
{
	uint4 c;
	c.x = (a.x * NIGHTCAP_FNV_PRIME ^ b.x) % 0xFFFFFFFF;
	c.y = (a.y * NIGHTCAP_FNV_PRIME ^ b.y) % 0xFFFFFFFF;
	c.z = (a.z * NIGHTCAP_FNV_PRIME ^ b.z) % 0xFFFFFFFF;
	c.w = (a.w * NIGHTCAP_FNV_PRIME ^ b.w) % 0xFFFFFFFF;
	return c;
}

__device__ __inline__ uint4 fnv4_int(uint4 a, uint32_t b)
{
	uint4 c;
	c.x = (a.x * NIGHTCAP_FNV_PRIME ^ b) % 0xFFFFFFFF;
	c.y = (a.y * NIGHTCAP_FNV_PRIME ^ b) % 0xFFFFFFFF;
	c.z = (a.z * NIGHTCAP_FNV_PRIME ^ b) % 0xFFFFFFFF;
	c.w = (a.w * NIGHTCAP_FNV_PRIME ^ b) % 0xFFFFFFFF;
	return c;
}

__device__ __inline__ uint32_t fnv_reduce(uint4 v)
{
	return fnv(fnv(fnv(v.x, v.y), v.z), v.w);
}

__device__ __inline__ void hashimoto_mix(uint32_t* headerHash, uint32_t* mixhash)
{
	NCMixNode mix;

	#pragma unroll 2
	for (int i=0; i<2; i++)
	{
		mix.values[i*8] = headerHash[0];
		mix.values[(i*8)+1] = headerHash[1];
		mix.values[(i*8)+2] = headerHash[2];
		mix.values[(i*8)+3] = headerHash[3];
		mix.values[(i*8)+4] = headerHash[4];
		mix.values[(i*8)+5] = headerHash[5];
		mix.values[(i*8)+6] = headerHash[6];
		mix.values[(i*8)+7] = headerHash[7];
	}

	uint32_t header_int = mix.values[0];

	for (uint32_t i = 0; i < 64; i++) {
		const uint32_t p = fnv(i ^ header_int, mix.values[i % 16]) % (nc_d_dag_size / 2);
		mix.nodes4[0] = fnv4(mix.nodes4[0], nc_d_dag[p].nodes4[0]);
		mix.nodes4[1] = fnv4(mix.nodes4[1], nc_d_dag[p].nodes4[1]);
		mix.nodes4[2] = fnv4(mix.nodes4[2], nc_d_dag[p].nodes4[2]);
		mix.nodes4[3] = fnv4(mix.nodes4[3], nc_d_dag[p].nodes4[3]);
	}

	mixhash[0] = fnv_reduce(mix.nodes4[0]);
	mixhash[1] = fnv_reduce(mix.nodes4[1]);
	mixhash[2] = fnv_reduce(mix.nodes4[2]);
	mixhash[3] = fnv_reduce(mix.nodes4[3]);
}

#define GSPREC(a,b,c,d,x,y) { \
	v[a] += (m[x] ^ u256[y]) + v[b]; \
	v[d] = __byte_perm(v[d] ^ v[a],0, 0x1032); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 12); \
	v[a] += (m[y] ^ u256[x]) + v[b]; \
	v[d] = __byte_perm(v[d] ^ v[a],0, 0x0321); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 7); \
						}

__device__ __inline__ uint2 ROR8(const uint2 a) {
	uint2 result;
	result.x = __byte_perm(a.y, a.x, 0x0765);
	result.y = __byte_perm(a.x, a.y, 0x0765);
	return result;
}


__constant__ uint2 keccak_round_constants35[24] = {
	{ 0x00000001ul, 0x00000000 }, { 0x00008082ul, 0x00000000 },
	{ 0x0000808aul, 0x80000000 }, { 0x80008000ul, 0x80000000 },
	{ 0x0000808bul, 0x00000000 }, { 0x80000001ul, 0x00000000 },
	{ 0x80008081ul, 0x80000000 }, { 0x00008009ul, 0x80000000 },
	{ 0x0000008aul, 0x00000000 }, { 0x00000088ul, 0x00000000 },
	{ 0x80008009ul, 0x00000000 }, { 0x8000000aul, 0x00000000 },
	{ 0x8000808bul, 0x00000000 }, { 0x0000008bul, 0x80000000 },
	{ 0x00008089ul, 0x80000000 }, { 0x00008003ul, 0x80000000 },
	{ 0x00008002ul, 0x80000000 }, { 0x00000080ul, 0x80000000 },
	{ 0x0000800aul, 0x00000000 }, { 0x8000000aul, 0x80000000 },
	{ 0x80008081ul, 0x80000000 }, { 0x00008080ul, 0x80000000 },
	{ 0x80000001ul, 0x00000000 }, { 0x80008008ul, 0x80000000 }
};

static void __forceinline__ __device__ keccak_block(uint2 *s)
{
	uint2 bc[5], tmpxor[5], u, v;
	//	uint2 s[25];

	#pragma unroll 1
	for (int i = 0; i < 24; i++)
	{
		#pragma unroll
		for (uint32_t x = 0; x < 5; x++)
			tmpxor[x] = s[x] ^ s[x + 5] ^ s[x + 10] ^ s[x + 15] ^ s[x + 20];

		bc[0] = tmpxor[0] ^ ROL2(tmpxor[2], 1);
		bc[1] = tmpxor[1] ^ ROL2(tmpxor[3], 1);
		bc[2] = tmpxor[2] ^ ROL2(tmpxor[4], 1);
		bc[3] = tmpxor[3] ^ ROL2(tmpxor[0], 1);
		bc[4] = tmpxor[4] ^ ROL2(tmpxor[1], 1);

		u = s[1] ^ bc[0];

		s[0] ^= bc[4];
		s[1] = ROL2(s[6] ^ bc[0], 44);
		s[6] = ROL2(s[9] ^ bc[3], 20);
		s[9] = ROL2(s[22] ^ bc[1], 61);
		s[22] = ROL2(s[14] ^ bc[3], 39);
		s[14] = ROL2(s[20] ^ bc[4], 18);
		s[20] = ROL2(s[2] ^ bc[1], 62);
		s[2] = ROL2(s[12] ^ bc[1], 43);
		s[12] = ROL2(s[13] ^ bc[2], 25);
		s[13] = ROL8(s[19] ^ bc[3]);
		s[19] = ROR8(s[23] ^ bc[2]);
		s[23] = ROL2(s[15] ^ bc[4], 41);
		s[15] = ROL2(s[4] ^ bc[3], 27);
		s[4] = ROL2(s[24] ^ bc[3], 14);
		s[24] = ROL2(s[21] ^ bc[0], 2);
		s[21] = ROL2(s[8] ^ bc[2], 55);
		s[8] = ROL2(s[16] ^ bc[0], 45);
		s[16] = ROL2(s[5] ^ bc[4], 36);
		s[5] = ROL2(s[3] ^ bc[2], 28);
		s[3] = ROL2(s[18] ^ bc[2], 21);
		s[18] = ROL2(s[17] ^ bc[1], 15);
		s[17] = ROL2(s[11] ^ bc[0], 10);
		s[11] = ROL2(s[7] ^ bc[1], 6);
		s[7] = ROL2(s[10] ^ bc[4], 3);
		s[10] = ROL2(u, 1);

		u = s[0]; v = s[1]; s[0] ^= (~v) & s[2]; s[1] ^= (~s[2]) & s[3]; s[2] ^= (~s[3]) & s[4]; s[3] ^= (~s[4]) & u; s[4] ^= (~u) & v;
		u = s[5]; v = s[6]; s[5] ^= (~v) & s[7]; s[6] ^= (~s[7]) & s[8]; s[7] ^= (~s[8]) & s[9]; s[8] ^= (~s[9]) & u; s[9] ^= (~u) & v;
		u = s[10]; v = s[11]; s[10] ^= (~v) & s[12]; s[11] ^= (~s[12]) & s[13]; s[12] ^= (~s[13]) & s[14]; s[13] ^= (~s[14]) & u; s[14] ^= (~u) & v;
		u = s[15]; v = s[16]; s[15] ^= (~v) & s[17]; s[16] ^= (~s[17]) & s[18]; s[17] ^= (~s[18]) & s[19]; s[18] ^= (~s[19]) & u; s[19] ^= (~u) & v;
		u = s[20]; v = s[21]; s[20] ^= (~v) & s[22]; s[21] ^= (~s[22]) & s[23]; s[22] ^= (~s[23]) & s[24]; s[23] ^= (~s[24]) & u; s[24] ^= (~u) & v;
		s[0] ^= keccak_round_constants35[i];
	}
}

//__launch_bounds__(256)
__global__
void nightcap_gpu_hash_52(const uint32_t threads, const uint32_t startNonce, uint64_t * Hash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t T0 = 0x1a0U;
		uint32_t v[16];

		const uint32_t  u256[16] = {
			0x243F6A88, 0x85A308D3,
			0x13198A2E, 0x03707344,
			0xA4093822, 0x299F31D0,
			0x082EFA98, 0xEC4E6C89,
			0x452821E6, 0x38D01377,
			0xBE5466CF, 0x34E90C6C,
			0xC0AC29B7, 0xC97C50DD,
			0x3F84D5B5, 0xB5470917
		};

		uint32_t m[16];

		LOHI(m[0], m[1], __ldg(&((uint64_t*)Hash)[thread]));
		LOHI(m[2], m[3], __ldg(&((uint64_t*)Hash)[thread + 1 * threads]));
		LOHI(m[4], m[5], __ldg(&((uint64_t*)Hash)[thread + 2 * threads]));
		LOHI(m[6], m[7], __ldg(&((uint64_t*)Hash)[thread + 3 * threads]));

		m[8] = nc_d_height;

		// mix mix mix
		hashimoto_mix(m, &m[9]);

		#pragma unroll 8
		for (uint32_t i=0; i<13; i++)
		{
			m[i] = cuda_swab32(m[i]);
		}

		// padding
		m[13] = 2147483649;
		m[14] = 0;
		m[15] = 416;

		v[0] = ((uint32_t)(0x6a09e667U)); 
		v[1] = ((uint32_t)(0xbb67ae85U)); 
		v[2] = ((uint32_t)(0x3c6ef372U)); 
		v[3] = ((uint32_t)(0xa54ff53aU)); 
		v[4] = ((uint32_t)(0x510e527fU)); 
		v[5] = ((uint32_t)(0x9b05688cU)); 
		v[6] = ((uint32_t)(0x1f83d9abU)); 
		v[7] = ((uint32_t)(0x5be0cd19U)); 

		v[8] = u256[0];
		v[9] = u256[1];
		v[10] = u256[2];
		v[11] = u256[3];
		v[12] = u256[4] ^ T0;
		v[13] = u256[5] ^ T0;
		v[14] = u256[6];
		v[15] = u256[7];

		//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
		GSPREC(0, 4, 0x8, 0xC, 0, 1);
		GSPREC(1, 5, 0x9, 0xD, 2, 3);
		GSPREC(2, 6, 0xA, 0xE, 4, 5);
		GSPREC(3, 7, 0xB, 0xF, 6, 7);
		GSPREC(0, 5, 0xA, 0xF, 8, 9);
		GSPREC(1, 6, 0xB, 0xC, 10, 11);
		GSPREC(2, 7, 0x8, 0xD, 12, 13);
		GSPREC(3, 4, 0x9, 0xE, 14, 15);
		//	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
		GSPREC(0, 4, 0x8, 0xC, 14, 10);
		GSPREC(1, 5, 0x9, 0xD, 4, 8);
		GSPREC(2, 6, 0xA, 0xE, 9, 15);
		GSPREC(3, 7, 0xB, 0xF, 13, 6);
		GSPREC(0, 5, 0xA, 0xF, 1, 12);
		GSPREC(1, 6, 0xB, 0xC, 0, 2);
		GSPREC(2, 7, 0x8, 0xD, 11, 7);
		GSPREC(3, 4, 0x9, 0xE, 5, 3);
		//	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
		GSPREC(0, 4, 0x8, 0xC, 11, 8);
		GSPREC(1, 5, 0x9, 0xD, 12, 0);
		GSPREC(2, 6, 0xA, 0xE, 5, 2);
		GSPREC(3, 7, 0xB, 0xF, 15, 13);
		GSPREC(0, 5, 0xA, 0xF, 10, 14);
		GSPREC(1, 6, 0xB, 0xC, 3, 6);
		GSPREC(2, 7, 0x8, 0xD, 7, 1);
		GSPREC(3, 4, 0x9, 0xE, 9, 4);
		//	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
		GSPREC(0, 4, 0x8, 0xC, 7, 9);
		GSPREC(1, 5, 0x9, 0xD, 3, 1);
		GSPREC(2, 6, 0xA, 0xE, 13, 12);
		GSPREC(3, 7, 0xB, 0xF, 11, 14);
		GSPREC(0, 5, 0xA, 0xF, 2, 6);
		GSPREC(1, 6, 0xB, 0xC, 5, 10);
		GSPREC(2, 7, 0x8, 0xD, 4, 0);
		GSPREC(3, 4, 0x9, 0xE, 15, 8);
		//	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
		GSPREC(0, 4, 0x8, 0xC, 9, 0);
		GSPREC(1, 5, 0x9, 0xD, 5, 7);
		GSPREC(2, 6, 0xA, 0xE, 2, 4);
		GSPREC(3, 7, 0xB, 0xF, 10, 15);
		GSPREC(0, 5, 0xA, 0xF, 14, 1);
		GSPREC(1, 6, 0xB, 0xC, 11, 12);
		GSPREC(2, 7, 0x8, 0xD, 6, 8);
		GSPREC(3, 4, 0x9, 0xE, 3, 13);
		//	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
		GSPREC(0, 4, 0x8, 0xC, 2, 12);
		GSPREC(1, 5, 0x9, 0xD, 6, 10);
		GSPREC(2, 6, 0xA, 0xE, 0, 11);
		GSPREC(3, 7, 0xB, 0xF, 8, 3);
		GSPREC(0, 5, 0xA, 0xF, 4, 13);
		GSPREC(1, 6, 0xB, 0xC, 7, 5);
		GSPREC(2, 7, 0x8, 0xD, 15, 14);
		GSPREC(3, 4, 0x9, 0xE, 1, 9);
		//	{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
		GSPREC(0, 4, 0x8, 0xC, 12, 5);
		GSPREC(1, 5, 0x9, 0xD, 1, 15);
		GSPREC(2, 6, 0xA, 0xE, 14, 13);
		GSPREC(3, 7, 0xB, 0xF, 4, 10);
		GSPREC(0, 5, 0xA, 0xF, 0, 7);
		GSPREC(1, 6, 0xB, 0xC, 6, 3);
		GSPREC(2, 7, 0x8, 0xD, 9, 2);
		GSPREC(3, 4, 0x9, 0xE, 8, 11);
		//	{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
		GSPREC(0, 4, 0x8, 0xC, 13, 11);
		GSPREC(1, 5, 0x9, 0xD, 7, 14);
		GSPREC(2, 6, 0xA, 0xE, 12, 1);
		GSPREC(3, 7, 0xB, 0xF, 3, 9);
		GSPREC(0, 5, 0xA, 0xF, 5, 0);
		GSPREC(1, 6, 0xB, 0xC, 15, 4);
		GSPREC(2, 7, 0x8, 0xD, 8, 6);
		GSPREC(3, 4, 0x9, 0xE, 2, 10);
		//	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
		GSPREC(0, 4, 0x8, 0xC, 6, 15);
		GSPREC(1, 5, 0x9, 0xD, 14, 9);
		GSPREC(2, 6, 0xA, 0xE, 11, 3);
		GSPREC(3, 7, 0xB, 0xF, 0, 8);
		GSPREC(0, 5, 0xA, 0xF, 12, 2);
		GSPREC(1, 6, 0xB, 0xC, 13, 7);
		GSPREC(2, 7, 0x8, 0xD, 1, 4);
		GSPREC(3, 4, 0x9, 0xE, 10, 5);
		//	{ 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
		GSPREC(0, 4, 0x8, 0xC, 10, 2);
		GSPREC(1, 5, 0x9, 0xD, 8, 4);
		GSPREC(2, 6, 0xA, 0xE, 7, 6);
		GSPREC(3, 7, 0xB, 0xF, 1, 5);
		GSPREC(0, 5, 0xA, 0xF, 15, 11);
		GSPREC(1, 6, 0xB, 0xC, 9, 14);
		GSPREC(2, 7, 0x8, 0xD, 3, 12);
		GSPREC(3, 4, 0x9, 0xE, 13, 0);
		//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
		GSPREC(0, 4, 0x8, 0xC, 0, 1);
		GSPREC(1, 5, 0x9, 0xD, 2, 3);
		GSPREC(2, 6, 0xA, 0xE, 4, 5);
		GSPREC(3, 7, 0xB, 0xF, 6, 7);
		GSPREC(0, 5, 0xA, 0xF, 8, 9);
		GSPREC(1, 6, 0xB, 0xC, 10, 11);
		GSPREC(2, 7, 0x8, 0xD, 12, 13);
		GSPREC(3, 4, 0x9, 0xE, 14, 15);
		//	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
		GSPREC(0, 4, 0x8, 0xC, 14, 10);
		GSPREC(1, 5, 0x9, 0xD, 4, 8);
		GSPREC(2, 6, 0xA, 0xE, 9, 15);
		GSPREC(3, 7, 0xB, 0xF, 13, 6);
		GSPREC(0, 5, 0xA, 0xF, 1, 12);
		GSPREC(1, 6, 0xB, 0xC, 0, 2);
		GSPREC(2, 7, 0x8, 0xD, 11, 7);
		GSPREC(3, 4, 0x9, 0xE, 5, 3);
		//	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
		GSPREC(0, 4, 0x8, 0xC, 11, 8);
		GSPREC(1, 5, 0x9, 0xD, 12, 0);
		GSPREC(2, 6, 0xA, 0xE, 5, 2);
		GSPREC(3, 7, 0xB, 0xF, 15, 13);
		GSPREC(0, 5, 0xA, 0xF, 10, 14);
		GSPREC(1, 6, 0xB, 0xC, 3, 6);
		GSPREC(2, 7, 0x8, 0xD, 7, 1);
		GSPREC(3, 4, 0x9, 0xE, 9, 4);
		//	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
		GSPREC(0, 4, 0x8, 0xC, 7, 9);
		GSPREC(1, 5, 0x9, 0xD, 3, 1);
		GSPREC(2, 6, 0xA, 0xE, 13, 12);
		GSPREC(3, 7, 0xB, 0xF, 11, 14);
		GSPREC(0, 5, 0xA, 0xF, 2, 6);
		GSPREC(1, 6, 0xB, 0xC, 5, 10);
		GSPREC(2, 7, 0x8, 0xD, 4, 0);
		GSPREC(3, 4, 0x9, 0xE, 15, 8);

		uint32_t h[8];

		h[0] = cuda_swab32(0x6a09e667U ^ v[0] ^ v[8]);
		h[1] = cuda_swab32(0xbb67ae85U ^ v[1] ^ v[9]);
		h[2] = cuda_swab32(0x3c6ef372U ^ v[2] ^ v[10]);
		h[3] = cuda_swab32(0xa54ff53aU ^ v[3] ^ v[11]);
		h[4] = cuda_swab32(0x510e527fU ^ v[4] ^ v[12]);
		h[5] = cuda_swab32(0x9b05688cU ^ v[5] ^ v[13]);
		h[6] = cuda_swab32(0x1f83d9abU ^ v[6] ^ v[14]);
		h[7] = cuda_swab32(0x5be0cd19U ^ v[7] ^ v[15]);

		uint2 keccak_gpu_state[25] = { 0 };
		keccak_gpu_state[0].x = h[0];
		keccak_gpu_state[0].y = h[1];
		keccak_gpu_state[1].x = h[2];
		keccak_gpu_state[1].y = h[3];
		keccak_gpu_state[2].x = h[4];
		keccak_gpu_state[2].y = h[5];
		keccak_gpu_state[3].x = h[6];
		keccak_gpu_state[3].y = h[7];
		keccak_gpu_state[4] = make_uint2(1, 0);

		keccak_gpu_state[16] = make_uint2(0, 0x80000000);
		keccak_block(keccak_gpu_state);

		uint64_t *outputHash = (uint64_t *)Hash;
		#pragma unroll 4
		for (int i = 0; i<4; i++)
			outputHash[i*threads + thread] = devectorize(keccak_gpu_state[i]);
	}
}


__host__
void nightcap_blakeKeccak_hashimoto_cpu_hash_32(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash, int order)
{
	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	nightcap_gpu_hash_52 <<<grid, block>>> (threads, startNonce, Hash);
}

__global__
void nightcap_recalc_dag_item(const uint32_t start)
{
	uint32_t NodeIdx = start + (blockIdx.x * blockDim.x + threadIdx.x);

	if (NodeIdx >= nc_d_dag_size)
	return;

	NCLightNode DAGNode = nc_d_light[NodeIdx % nc_d_light_size];

	DAGNode.values[0] ^= NodeIdx;

	const uint32_t T0 = ((0xFFFFFE00) + 256) + 512;
	const uint32_t  u256[16] = {
		0x243F6A88, 0x85A308D3,
		0x13198A2E, 0x03707344,
		0xA4093822, 0x299F31D0,
		0x082EFA98, 0xEC4E6C89,
		0x452821E6, 0x38D01377,
		0xBE5466CF, 0x34E90C6C,
		0xC0AC29B7, 0xC97C50DD,
		0x3F84D5B5, 0xB5470917
	};

	// First blake round
	{
		uint32_t v[16];
		uint32_t m[16];

		#pragma unroll 8
		for (int i = 0; i < 8; i++)
		{
			m[i] = cuda_swab32(DAGNode.values[i]);
		}

		// padding
		m[8] = 2147483648;
		m[9] = 0;
		m[10] = 0;
		m[11] = 0;
		m[12] = 0;
		m[13] = 1;
		m[14] = 0;
		m[15] = 256;

		v[0] = ((uint32_t)(0x6a09e667U)); 
		v[1] = ((uint32_t)(0xbb67ae85U)); 
		v[2] = ((uint32_t)(0x3c6ef372U)); 
		v[3] = ((uint32_t)(0xa54ff53aU)); 
		v[4] = ((uint32_t)(0x510e527fU)); 
		v[5] = ((uint32_t)(0x9b05688cU)); 
		v[6] = ((uint32_t)(0x1f83d9abU)); 
		v[7] = ((uint32_t)(0x5be0cd19U)); 

		v[8] = u256[0];
		v[9] = u256[1];
		v[10] = u256[2];
		v[11] = u256[3];
		v[12] = u256[4] ^ T0;
		v[13] = u256[5] ^ T0;
		v[14] = u256[6];
		v[15] = u256[7];

		//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
		GSPREC(0, 4, 0x8, 0xC, 0, 1);
		GSPREC(1, 5, 0x9, 0xD, 2, 3);
		GSPREC(2, 6, 0xA, 0xE, 4, 5);
		GSPREC(3, 7, 0xB, 0xF, 6, 7);
		GSPREC(0, 5, 0xA, 0xF, 8, 9);
		GSPREC(1, 6, 0xB, 0xC, 10, 11);
		GSPREC(2, 7, 0x8, 0xD, 12, 13);
		GSPREC(3, 4, 0x9, 0xE, 14, 15);
		//	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
		GSPREC(0, 4, 0x8, 0xC, 14, 10);
		GSPREC(1, 5, 0x9, 0xD, 4, 8);
		GSPREC(2, 6, 0xA, 0xE, 9, 15);
		GSPREC(3, 7, 0xB, 0xF, 13, 6);
		GSPREC(0, 5, 0xA, 0xF, 1, 12);
		GSPREC(1, 6, 0xB, 0xC, 0, 2);
		GSPREC(2, 7, 0x8, 0xD, 11, 7);
		GSPREC(3, 4, 0x9, 0xE, 5, 3);
		//	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
		GSPREC(0, 4, 0x8, 0xC, 11, 8);
		GSPREC(1, 5, 0x9, 0xD, 12, 0);
		GSPREC(2, 6, 0xA, 0xE, 5, 2);
		GSPREC(3, 7, 0xB, 0xF, 15, 13);
		GSPREC(0, 5, 0xA, 0xF, 10, 14);
		GSPREC(1, 6, 0xB, 0xC, 3, 6);
		GSPREC(2, 7, 0x8, 0xD, 7, 1);
		GSPREC(3, 4, 0x9, 0xE, 9, 4);
		//	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
		GSPREC(0, 4, 0x8, 0xC, 7, 9);
		GSPREC(1, 5, 0x9, 0xD, 3, 1);
		GSPREC(2, 6, 0xA, 0xE, 13, 12);
		GSPREC(3, 7, 0xB, 0xF, 11, 14);
		GSPREC(0, 5, 0xA, 0xF, 2, 6);
		GSPREC(1, 6, 0xB, 0xC, 5, 10);
		GSPREC(2, 7, 0x8, 0xD, 4, 0);
		GSPREC(3, 4, 0x9, 0xE, 15, 8);
		//	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
		GSPREC(0, 4, 0x8, 0xC, 9, 0);
		GSPREC(1, 5, 0x9, 0xD, 5, 7);
		GSPREC(2, 6, 0xA, 0xE, 2, 4);
		GSPREC(3, 7, 0xB, 0xF, 10, 15);
		GSPREC(0, 5, 0xA, 0xF, 14, 1);
		GSPREC(1, 6, 0xB, 0xC, 11, 12);
		GSPREC(2, 7, 0x8, 0xD, 6, 8);
		GSPREC(3, 4, 0x9, 0xE, 3, 13);
		//	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
		GSPREC(0, 4, 0x8, 0xC, 2, 12);
		GSPREC(1, 5, 0x9, 0xD, 6, 10);
		GSPREC(2, 6, 0xA, 0xE, 0, 11);
		GSPREC(3, 7, 0xB, 0xF, 8, 3);
		GSPREC(0, 5, 0xA, 0xF, 4, 13);
		GSPREC(1, 6, 0xB, 0xC, 7, 5);
		GSPREC(2, 7, 0x8, 0xD, 15, 14);
		GSPREC(3, 4, 0x9, 0xE, 1, 9);
		//	{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
		GSPREC(0, 4, 0x8, 0xC, 12, 5);
		GSPREC(1, 5, 0x9, 0xD, 1, 15);
		GSPREC(2, 6, 0xA, 0xE, 14, 13);
		GSPREC(3, 7, 0xB, 0xF, 4, 10);
		GSPREC(0, 5, 0xA, 0xF, 0, 7);
		GSPREC(1, 6, 0xB, 0xC, 6, 3);
		GSPREC(2, 7, 0x8, 0xD, 9, 2);
		GSPREC(3, 4, 0x9, 0xE, 8, 11);
		//	{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
		GSPREC(0, 4, 0x8, 0xC, 13, 11);
		GSPREC(1, 5, 0x9, 0xD, 7, 14);
		GSPREC(2, 6, 0xA, 0xE, 12, 1);
		GSPREC(3, 7, 0xB, 0xF, 3, 9);
		GSPREC(0, 5, 0xA, 0xF, 5, 0);
		GSPREC(1, 6, 0xB, 0xC, 15, 4);
		GSPREC(2, 7, 0x8, 0xD, 8, 6);
		GSPREC(3, 4, 0x9, 0xE, 2, 10);
		//	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
		GSPREC(0, 4, 0x8, 0xC, 6, 15);
		GSPREC(1, 5, 0x9, 0xD, 14, 9);
		GSPREC(2, 6, 0xA, 0xE, 11, 3);
		GSPREC(3, 7, 0xB, 0xF, 0, 8);
		GSPREC(0, 5, 0xA, 0xF, 12, 2);
		GSPREC(1, 6, 0xB, 0xC, 13, 7);
		GSPREC(2, 7, 0x8, 0xD, 1, 4);
		GSPREC(3, 4, 0x9, 0xE, 10, 5);
		//	{ 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
		GSPREC(0, 4, 0x8, 0xC, 10, 2);
		GSPREC(1, 5, 0x9, 0xD, 8, 4);
		GSPREC(2, 6, 0xA, 0xE, 7, 6);
		GSPREC(3, 7, 0xB, 0xF, 1, 5);
		GSPREC(0, 5, 0xA, 0xF, 15, 11);
		GSPREC(1, 6, 0xB, 0xC, 9, 14);
		GSPREC(2, 7, 0x8, 0xD, 3, 12);
		GSPREC(3, 4, 0x9, 0xE, 13, 0);
		//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
		GSPREC(0, 4, 0x8, 0xC, 0, 1);
		GSPREC(1, 5, 0x9, 0xD, 2, 3);
		GSPREC(2, 6, 0xA, 0xE, 4, 5);
		GSPREC(3, 7, 0xB, 0xF, 6, 7);
		GSPREC(0, 5, 0xA, 0xF, 8, 9);
		GSPREC(1, 6, 0xB, 0xC, 10, 11);
		GSPREC(2, 7, 0x8, 0xD, 12, 13);
		GSPREC(3, 4, 0x9, 0xE, 14, 15);
		//	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
		GSPREC(0, 4, 0x8, 0xC, 14, 10);
		GSPREC(1, 5, 0x9, 0xD, 4, 8);
		GSPREC(2, 6, 0xA, 0xE, 9, 15);
		GSPREC(3, 7, 0xB, 0xF, 13, 6);
		GSPREC(0, 5, 0xA, 0xF, 1, 12);
		GSPREC(1, 6, 0xB, 0xC, 0, 2);
		GSPREC(2, 7, 0x8, 0xD, 11, 7);
		GSPREC(3, 4, 0x9, 0xE, 5, 3);
		//	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
		GSPREC(0, 4, 0x8, 0xC, 11, 8);
		GSPREC(1, 5, 0x9, 0xD, 12, 0);
		GSPREC(2, 6, 0xA, 0xE, 5, 2);
		GSPREC(3, 7, 0xB, 0xF, 15, 13);
		GSPREC(0, 5, 0xA, 0xF, 10, 14);
		GSPREC(1, 6, 0xB, 0xC, 3, 6);
		GSPREC(2, 7, 0x8, 0xD, 7, 1);
		GSPREC(3, 4, 0x9, 0xE, 9, 4);
		//	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
		GSPREC(0, 4, 0x8, 0xC, 7, 9);
		GSPREC(1, 5, 0x9, 0xD, 3, 1);
		GSPREC(2, 6, 0xA, 0xE, 13, 12);
		GSPREC(3, 7, 0xB, 0xF, 11, 14);
		GSPREC(0, 5, 0xA, 0xF, 2, 6);
		GSPREC(1, 6, 0xB, 0xC, 5, 10);
		GSPREC(2, 7, 0x8, 0xD, 4, 0);
		GSPREC(3, 4, 0x9, 0xE, 15, 8);

		DAGNode.values[0] = cuda_swab32(0x6a09e667U ^ v[0] ^ v[8]);
		DAGNode.values[1] = cuda_swab32(0xbb67ae85U ^ v[1] ^ v[9]);
		DAGNode.values[2] = cuda_swab32(0x3c6ef372U ^ v[2] ^ v[10]);
		DAGNode.values[3] = cuda_swab32(0xa54ff53aU ^ v[3] ^ v[11]);
		DAGNode.values[4] = cuda_swab32(0x510e527fU ^ v[4] ^ v[12]);
		DAGNode.values[5] = cuda_swab32(0x9b05688cU ^ v[5] ^ v[13]);
		DAGNode.values[6] = cuda_swab32(0x1f83d9abU ^ v[6] ^ v[14]);
		DAGNode.values[7] = cuda_swab32(0x5be0cd19U ^ v[7] ^ v[15]);
	}

	for (uint32_t parent = 0; parent < NIGHTCAP_DATASET_PARENTS; ++parent)
	{
		// Calculate parent
		uint32_t ParentIdx = fnv(NodeIdx ^ parent, DAGNode.values[parent & 7]) % nc_d_light_size;
		const NCLightNode *ParentNode = nc_d_light + ParentIdx;

		DAGNode.nodes4[0] = fnv4_int(DAGNode.nodes4[0], ParentNode->values[0]);
		DAGNode.nodes4[1] = fnv4_int(DAGNode.nodes4[1], ParentNode->values[0]);
	}

	
	// Last blake round
	{
		uint32_t v[16];
		uint32_t m[16];

		#pragma unroll 8
		for (int i = 0; i < 8; i++)
		{
			m[i] = cuda_swab32(DAGNode.values[i]);
		}

		// padding
		m[8] = 2147483648;
		m[9] = 0;
		m[10] = 0;
		m[11] = 0;
		m[12] = 0;
		m[13] = 1;
		m[14] = 0;
		m[15] = 256;

		v[0] = ((uint32_t)(0x6a09e667U)); 
		v[1] = ((uint32_t)(0xbb67ae85U)); 
		v[2] = ((uint32_t)(0x3c6ef372U)); 
		v[3] = ((uint32_t)(0xa54ff53aU)); 
		v[4] = ((uint32_t)(0x510e527fU)); 
		v[5] = ((uint32_t)(0x9b05688cU)); 
		v[6] = ((uint32_t)(0x1f83d9abU)); 
		v[7] = ((uint32_t)(0x5be0cd19U)); 

		v[8] = u256[0];
		v[9] = u256[1];
		v[10] = u256[2];
		v[11] = u256[3];
		v[12] = u256[4] ^ T0;
		v[13] = u256[5] ^ T0;
		v[14] = u256[6];
		v[15] = u256[7];

		//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
		GSPREC(0, 4, 0x8, 0xC, 0, 1);
		GSPREC(1, 5, 0x9, 0xD, 2, 3);
		GSPREC(2, 6, 0xA, 0xE, 4, 5);
		GSPREC(3, 7, 0xB, 0xF, 6, 7);
		GSPREC(0, 5, 0xA, 0xF, 8, 9);
		GSPREC(1, 6, 0xB, 0xC, 10, 11);
		GSPREC(2, 7, 0x8, 0xD, 12, 13);
		GSPREC(3, 4, 0x9, 0xE, 14, 15);
		//	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
		GSPREC(0, 4, 0x8, 0xC, 14, 10);
		GSPREC(1, 5, 0x9, 0xD, 4, 8);
		GSPREC(2, 6, 0xA, 0xE, 9, 15);
		GSPREC(3, 7, 0xB, 0xF, 13, 6);
		GSPREC(0, 5, 0xA, 0xF, 1, 12);
		GSPREC(1, 6, 0xB, 0xC, 0, 2);
		GSPREC(2, 7, 0x8, 0xD, 11, 7);
		GSPREC(3, 4, 0x9, 0xE, 5, 3);
		//	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
		GSPREC(0, 4, 0x8, 0xC, 11, 8);
		GSPREC(1, 5, 0x9, 0xD, 12, 0);
		GSPREC(2, 6, 0xA, 0xE, 5, 2);
		GSPREC(3, 7, 0xB, 0xF, 15, 13);
		GSPREC(0, 5, 0xA, 0xF, 10, 14);
		GSPREC(1, 6, 0xB, 0xC, 3, 6);
		GSPREC(2, 7, 0x8, 0xD, 7, 1);
		GSPREC(3, 4, 0x9, 0xE, 9, 4);
		//	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
		GSPREC(0, 4, 0x8, 0xC, 7, 9);
		GSPREC(1, 5, 0x9, 0xD, 3, 1);
		GSPREC(2, 6, 0xA, 0xE, 13, 12);
		GSPREC(3, 7, 0xB, 0xF, 11, 14);
		GSPREC(0, 5, 0xA, 0xF, 2, 6);
		GSPREC(1, 6, 0xB, 0xC, 5, 10);
		GSPREC(2, 7, 0x8, 0xD, 4, 0);
		GSPREC(3, 4, 0x9, 0xE, 15, 8);
		//	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
		GSPREC(0, 4, 0x8, 0xC, 9, 0);
		GSPREC(1, 5, 0x9, 0xD, 5, 7);
		GSPREC(2, 6, 0xA, 0xE, 2, 4);
		GSPREC(3, 7, 0xB, 0xF, 10, 15);
		GSPREC(0, 5, 0xA, 0xF, 14, 1);
		GSPREC(1, 6, 0xB, 0xC, 11, 12);
		GSPREC(2, 7, 0x8, 0xD, 6, 8);
		GSPREC(3, 4, 0x9, 0xE, 3, 13);
		//	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
		GSPREC(0, 4, 0x8, 0xC, 2, 12);
		GSPREC(1, 5, 0x9, 0xD, 6, 10);
		GSPREC(2, 6, 0xA, 0xE, 0, 11);
		GSPREC(3, 7, 0xB, 0xF, 8, 3);
		GSPREC(0, 5, 0xA, 0xF, 4, 13);
		GSPREC(1, 6, 0xB, 0xC, 7, 5);
		GSPREC(2, 7, 0x8, 0xD, 15, 14);
		GSPREC(3, 4, 0x9, 0xE, 1, 9);
		//	{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
		GSPREC(0, 4, 0x8, 0xC, 12, 5);
		GSPREC(1, 5, 0x9, 0xD, 1, 15);
		GSPREC(2, 6, 0xA, 0xE, 14, 13);
		GSPREC(3, 7, 0xB, 0xF, 4, 10);
		GSPREC(0, 5, 0xA, 0xF, 0, 7);
		GSPREC(1, 6, 0xB, 0xC, 6, 3);
		GSPREC(2, 7, 0x8, 0xD, 9, 2);
		GSPREC(3, 4, 0x9, 0xE, 8, 11);
		//	{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
		GSPREC(0, 4, 0x8, 0xC, 13, 11);
		GSPREC(1, 5, 0x9, 0xD, 7, 14);
		GSPREC(2, 6, 0xA, 0xE, 12, 1);
		GSPREC(3, 7, 0xB, 0xF, 3, 9);
		GSPREC(0, 5, 0xA, 0xF, 5, 0);
		GSPREC(1, 6, 0xB, 0xC, 15, 4);
		GSPREC(2, 7, 0x8, 0xD, 8, 6);
		GSPREC(3, 4, 0x9, 0xE, 2, 10);
		//	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
		GSPREC(0, 4, 0x8, 0xC, 6, 15);
		GSPREC(1, 5, 0x9, 0xD, 14, 9);
		GSPREC(2, 6, 0xA, 0xE, 11, 3);
		GSPREC(3, 7, 0xB, 0xF, 0, 8);
		GSPREC(0, 5, 0xA, 0xF, 12, 2);
		GSPREC(1, 6, 0xB, 0xC, 13, 7);
		GSPREC(2, 7, 0x8, 0xD, 1, 4);
		GSPREC(3, 4, 0x9, 0xE, 10, 5);
		//	{ 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
		GSPREC(0, 4, 0x8, 0xC, 10, 2);
		GSPREC(1, 5, 0x9, 0xD, 8, 4);
		GSPREC(2, 6, 0xA, 0xE, 7, 6);
		GSPREC(3, 7, 0xB, 0xF, 1, 5);
		GSPREC(0, 5, 0xA, 0xF, 15, 11);
		GSPREC(1, 6, 0xB, 0xC, 9, 14);
		GSPREC(2, 7, 0x8, 0xD, 3, 12);
		GSPREC(3, 4, 0x9, 0xE, 13, 0);
		//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
		GSPREC(0, 4, 0x8, 0xC, 0, 1);
		GSPREC(1, 5, 0x9, 0xD, 2, 3);
		GSPREC(2, 6, 0xA, 0xE, 4, 5);
		GSPREC(3, 7, 0xB, 0xF, 6, 7);
		GSPREC(0, 5, 0xA, 0xF, 8, 9);
		GSPREC(1, 6, 0xB, 0xC, 10, 11);
		GSPREC(2, 7, 0x8, 0xD, 12, 13);
		GSPREC(3, 4, 0x9, 0xE, 14, 15);
		//	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
		GSPREC(0, 4, 0x8, 0xC, 14, 10);
		GSPREC(1, 5, 0x9, 0xD, 4, 8);
		GSPREC(2, 6, 0xA, 0xE, 9, 15);
		GSPREC(3, 7, 0xB, 0xF, 13, 6);
		GSPREC(0, 5, 0xA, 0xF, 1, 12);
		GSPREC(1, 6, 0xB, 0xC, 0, 2);
		GSPREC(2, 7, 0x8, 0xD, 11, 7);
		GSPREC(3, 4, 0x9, 0xE, 5, 3);
		//	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
		GSPREC(0, 4, 0x8, 0xC, 11, 8);
		GSPREC(1, 5, 0x9, 0xD, 12, 0);
		GSPREC(2, 6, 0xA, 0xE, 5, 2);
		GSPREC(3, 7, 0xB, 0xF, 15, 13);
		GSPREC(0, 5, 0xA, 0xF, 10, 14);
		GSPREC(1, 6, 0xB, 0xC, 3, 6);
		GSPREC(2, 7, 0x8, 0xD, 7, 1);
		GSPREC(3, 4, 0x9, 0xE, 9, 4);
		//	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
		GSPREC(0, 4, 0x8, 0xC, 7, 9);
		GSPREC(1, 5, 0x9, 0xD, 3, 1);
		GSPREC(2, 6, 0xA, 0xE, 13, 12);
		GSPREC(3, 7, 0xB, 0xF, 11, 14);
		GSPREC(0, 5, 0xA, 0xF, 2, 6);
		GSPREC(1, 6, 0xB, 0xC, 5, 10);
		GSPREC(2, 7, 0x8, 0xD, 4, 0);
		GSPREC(3, 4, 0x9, 0xE, 15, 8);

		DAGNode.values[0] = cuda_swab32(0x6a09e667U ^ v[0] ^ v[8]);
		DAGNode.values[1] = cuda_swab32(0xbb67ae85U ^ v[1] ^ v[9]);
		DAGNode.values[2] = cuda_swab32(0x3c6ef372U ^ v[2] ^ v[10]);
		DAGNode.values[3] = cuda_swab32(0xa54ff53aU ^ v[3] ^ v[11]);
		DAGNode.values[4] = cuda_swab32(0x510e527fU ^ v[4] ^ v[12]);
		DAGNode.values[5] = cuda_swab32(0x9b05688cU ^ v[5] ^ v[13]);
		DAGNode.values[6] = cuda_swab32(0x1f83d9abU ^ v[6] ^ v[14]);
		DAGNode.values[7] = cuda_swab32(0x5be0cd19U ^ v[7] ^ v[15]);
	}

	((NCLightNode*)nc_d_dag)[NodeIdx] = DAGNode;
}

__host__
void nightcap_recalc_dag(
	uint64_t dag_nodes,
	uint32_t threads
	)
{
	const uint32_t threadsperblock = 256;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	uint32_t const work = grid.x * block.x;
	uint32_t const fullRuns = dag_nodes / (work);
	uint32_t const restWork = dag_nodes % (work);

	// Normal runs
	for (uint32_t i = 0; i < fullRuns; i++)
	{
		//printf("NC RUN: %u\n", i * work);
		nightcap_recalc_dag_item <<<grid, block, 0>>>(i * work);

	}
	hipDeviceSynchronize();

	// Final run
	if (restWork > 0)
	{
	//printf("NC FINAL RUN: %u\n", fullRuns * work);
		nightcap_recalc_dag_item <<<grid, block, 0>>>(fullRuns * work);
	}

	hipDeviceSynchronize();

	hipGetLastError();
}



__host__
void nightcap_set_mix_constants(
	NCMixNode* _dag,
	uint32_t _dag_size,
	NCLightNode * _light,
	uint32_t _light_size,
	uint32_t _height
	)
{
	hipMemcpyToSymbol(HIP_SYMBOL(nc_d_dag), &_dag, sizeof(NCMixNode *));
	hipMemcpyToSymbol(HIP_SYMBOL(nc_d_dag_size), &_dag_size, sizeof(uint32_t));
	hipMemcpyToSymbol(HIP_SYMBOL(nc_d_light), &_light, sizeof(NCLightNode *));
	hipMemcpyToSymbol(HIP_SYMBOL(nc_d_light_size), &_light_size, sizeof(uint32_t));
	hipMemcpyToSymbol(HIP_SYMBOL(nc_d_height), &_height, sizeof(uint32_t));
}


static bool init[MAX_GPUS] = { 0 };

static bool ncDAGInit = false;

__host__
void nightcap_dag_update(int thr_id, int dev_id, uint32_t height, uint32_t throughput)
{
	uint32_t epoch = height / 400;
	NCGPUState* gpu_state = &nc_gpu_state[dev_id];
	NCThreadState* thread_state = &nc_thread_state[thr_id];

	applog(LOG_INFO, "nightcap_dag_update T%u gpu_state %x", thr_id, gpu_state);

	nightcap_dag_update_lock(dev_id);
	hipSetDevice(dev_id);
	//hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
	//CUDA_LOG_ERROR();

	//applog(LOG_INFO, "nightcap_dag_update got dag lock");


	// Regen light cache if required
	uint32_t** cache_ptr = nightcap_lock_cache(epoch);
	uint32_t cache_size = nightcap_get_cache_size(height);
	if (!*cache_ptr || **cache_ptr != epoch)
	{
		// Regenerate light cache
		unsigned char seedhash[32];
		memset(seedhash, '\0', sizeof(seedhash));
		sph_blake256_context ctx_blake;
		for (uint32_t i = 0; i < epoch; i++) {
			sph_blake256_init(&ctx_blake);
			sph_blake256(&ctx_blake, seedhash, 32);
			sph_blake256_close(&ctx_blake, seedhash);
		}

		if (*cache_ptr)
			free(*cache_ptr);
		*cache_ptr = (uint32_t*)malloc(cache_size + sizeof(uint32_t));
		nightcap_generate_cache((uint32_t*)((*cache_ptr)+1), seedhash, cache_size);

		(*cache_ptr)[0] = epoch;
	}
	
	gpu_state->cache_size = cache_size;
	gpu_state->num_cache_nodes = gpu_state->cache_size / 32;

	//applog(LOG_INFO, "nightcap_dag_update got cache lock");

	// epoch we were working on isn't the current epoch?!
	if (thread_state->epoch != epoch)
	{
		//applog(LOG_INFO, "nightcap_dag_update epoch mismatch");
		// We'll need to update this thread state
	    init[thr_id] = false;
	    thread_state->epoch = epoch;

		// Regen if we are the first thread to init or set the current epoch
		if ((gpu_state->epoch != epoch && epoch > gpu_state->epoch) || !ncDAGInit)
		{
			uint32_t dag_size = nightcap_get_full_size(height);
			applog(LOG_INFO, "Regenerating dag, waiting for device %s to be ready...", device_name[dev_id]);
			ncDAGInit = true;

			// Force other threads to finish and block in this function
			restart_threads();
			hipDeviceSynchronize();

			applog(LOG_INFO, "Device ready, regenerating dag for epoch %u...", epoch);

			gpu_state->dag_size = dag_size;
			gpu_state->num_dag_nodes = dag_size / 32;
			gpu_state->epoch = epoch;
			gpu_state->height = height;

			hipDeviceReset();
			applog(LOG_INFO, "G%u device reset", dev_id);
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);

			CUDA_SAFE_CALL(hipMalloc(reinterpret_cast<void**>(&gpu_state->cache_nodes), gpu_state->cache_size));
			CUDA_SAFE_CALL(hipMalloc(reinterpret_cast<void**>(&gpu_state->dag_nodes), gpu_state->dag_size));

			// Copy over cache
			CUDA_SAFE_CALL(hipMemcpy(gpu_state->cache_nodes, ((*cache_ptr)+1), gpu_state->cache_size, hipMemcpyHostToDevice));

			hipDeviceSynchronize();
			nightcap_unlock_cache(epoch); // free for other threads

			// Set node memory
			nightcap_set_mix_constants(gpu_state->dag_nodes, 
			                           gpu_state->num_dag_nodes,
			                           gpu_state->cache_nodes,
			                           gpu_state->num_cache_nodes,
			                           height);

			// Set blake stuff for dag gen
			blake256_cpu_init(thr_id, throughput);
			cuda_get_arch(thr_id);

			// Regen dag
			nightcap_recalc_dag(gpu_state->num_dag_nodes, 1UL << 21);
			applog(LOG_INFO, "T%u Dag regeneration complete", thr_id);

#ifdef NIGHTCAP_DEBUG_DAG
			// dump dag
			{
				uint32_t* dag_tmp = (uint32_t*)malloc(gpu_state->dag_size);
				CUDA_SAFE_CALL(hipMemcpy(dag_tmp, gpu_state->dag_nodes, gpu_state->dag_size, hipMemcpyDeviceToHost));

				FILE* fp = fopen("dag.dat", "wb");
				fwrite(dag_tmp, 1, gpu_state->dag_size, fp);
				fclose(fp);
				free(dag_tmp);
			}
#endif

			nightcap_dag_update_unlock(dev_id);

			applog(LOG_INFO, "T%u nightcap_dag_update returned", thr_id);
			return;
		}
	}
	

	// If we aren't init'd, init our cloverhash values
	if (!init[thr_id] || gpu_state->height != height)
	{
		applog(LOG_INFO, "T%u Using pregen dag with height %u.", thr_id, height);
		
		gpu_state->height = height;

		// Set node memory
		nightcap_set_mix_constants(gpu_state->dag_nodes, 
		                           gpu_state->num_dag_nodes,
		                           gpu_state->cache_nodes,
		                           gpu_state->num_cache_nodes,
		                           gpu_state->height);
	}
	nightcap_unlock_cache(epoch); // free for other threads

	nightcap_dag_update_unlock(dev_id);
	applog(LOG_INFO, "T%u nightcap_dag_update return NOUPDATE", thr_id);
}

//#define CLOVERHASH_COMPARE_ON_CPU


extern "C" int scanhash_nightcap(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	int dev_id = device_map[thr_id];
	int intensity = (device_sm[dev_id] < 500) ? 18 : is_windows() ? 19 : 20;
	if (strstr(device_name[dev_id], "GTX 10")) intensity = 20;
	uint32_t throughput = cuda_default_throughput(dev_id, 1UL << intensity);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	// Mutexes for gpu states need to be init'd
	nightcap_ensure_setup();

	if (opt_benchmark)
		ptarget[7] = 0x000f;
		
	//work->height = 25335; // HACK DEBUG

	// Make sure dag is up to date!
	nightcap_dag_update(thr_id, dev_id, work->height, throughput);

	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (!init[thr_id])
	{
		applog(LOG_DEBUG, "reinit thread %u", thr_id);
		// NOTE: nightcap_dag_update resets device

		size_t matrix_sz = 16 * sizeof(uint64_t) * 4 * 3;
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		blake256_cpu_init(thr_id, throughput);
		skein256_cpu_init(thr_id, throughput);
		bmw256_cpu_init(thr_id, throughput);

		cuda_get_arch(thr_id); // cuda_arch[] also used in cubehash256

		// SM 3 implentation requires a bit more memory
		if (device_sm[dev_id] < 500 || cuda_arch[dev_id] < 500)
			matrix_sz = 16 * sizeof(uint64_t) * 4 * 4;
			
		CUDA_SAFE_CALL(hipMalloc(&d_matrix[thr_id], matrix_sz * throughput));
		lyra2v2_cpu_init(thr_id, throughput, d_matrix[thr_id]);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], (size_t)32 * throughput));

		api_set_throughput(thr_id, throughput);
		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	//memset(pdata, '\0', 80); // DEBUG
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);


	blake256_cpu_setBlock_80(pdata);
	bmw256_setTarget(ptarget);

	#ifdef CLOVERHASH_COMPARE_ON_CPU
	uint64_t* temp_hashes = (uint64_t*)malloc((size_t)32 * throughput);
	#endif

	do {
		int order = 0;
		memset(work->nonces, 0, sizeof(work->nonces));

		// DEBUG uint64_t* temp_hashes = (uint64_t*)malloc((size_t)32 * throughput);

		// first pass
		blakeKeccak256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		cubehash256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		lyra2v2_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		skein256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		cubehash256_cpu_hash_32(thr_id, throughput,pdata[19], d_hash[thr_id], order++);
		bmw256_cpu_hash_32_to_hash(thr_id, throughput, pdata[19], d_hash[thr_id], order++);

		// mix
		nightcap_blakeKeccak_hashimoto_cpu_hash_32(thr_id, throughput,pdata[19], d_hash[thr_id], order++);



		// DEBUG
		/*
		hipDeviceSynchronize();
		CUDA_SAFE_CALL(hipMemcpy(temp_hashes, d_hash[thr_id], (size_t)32 * throughput, hipMemcpyDeviceToHost));

		for (uint32_t i=0; i<throughput; i++)
		{
			uint64_t hash[4];
			hash[0] = temp_hashes[i];
			hash[1] = temp_hashes[i + 1 * throughput];
			hash[2] = temp_hashes[i + 2 * throughput];
			hash[3] = temp_hashes[i + 3 * throughput];

			uint32_t* hash32 = (uint32_t*)hash;
			printf("Hashmimoto[%i] == %08x,%08x,%08x,%08x,%08x,%08x,%08x,%08x\n", i, hash32[0], hash32[1], hash32[2], hash32[3],hash32[4], hash32[5], hash32[6], hash32[7]);
		}
		exit(1);
		*/
		// END DEBUG


		// second pass
		cubehash256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		lyra2v2_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		skein256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		cubehash256_cpu_hash_32(thr_id, throughput,pdata[19], d_hash[thr_id], order++);


		#ifdef CLOVERHASH_COMPARE_ON_CPU
		bmw256_cpu_hash_32_to_hash(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		CUDA_SAFE_CALL(hipMemcpy(temp_hashes, d_hash[thr_id], (size_t)32 * throughput, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();

		work->nonces[0] = 0;
		work->nonces[1] = 0;

		for (uint32_t i=0; i<throughput; i++)
		{
			uint64_t hash[4];
			hash[0] = temp_hashes[i];
			hash[1] = temp_hashes[i + 1 * throughput];
			hash[2] = temp_hashes[i + 2 * throughput];
			hash[3] = temp_hashes[i + 3 * throughput];
			uint32_t* vhash = (uint32_t*)&hash[0];
			const uint32_t Htarg = ptarget[7];
			if (vhash[7] <= Htarg && fulltest(vhash, ptarget))
			{
				work->nonces[0] = pdata[19] + i;
				applog(LOG_INFO, "T%u found nonce %u on cpu in slot %i", thr_id, work->nonces[0], i);
				break;
			}
		}

		#else
		bmw256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], work->nonces);
		#endif

		/* DEBUG OUT
		bmw256_cpu_hash_32_to_hash(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		CUDA_SAFE_CALL(hipMemcpy(temp_hashes, d_hash[thr_id], (size_t)32 * throughput, hipMemcpyDeviceToHost));

		FILE* fp = fopen("out_hashes.dat", "wb");
		fwrite(temp_hashes, 1, (size_t)32 * throughput, fp);
		fclose(fp);

		for (uint32_t i=0; i<throughput; i++)
		{
			uint64_t hash[4];
			hash[0] = temp_hashes[i];
			hash[1] = temp_hashes[i + 1 * throughput];
			hash[2] = temp_hashes[i + 2 * throughput];
			hash[3] = temp_hashes[i + 3 * throughput];

			uint32_t* hash32 = (uint32_t*)hash;
			printf("Hashmimoto[%i] == %08x,%08x,%08x,%08x,%08x,%08x,%08x,%08x\n", i, hash32[0], hash32[1], hash32[2], hash32[3],hash32[4], hash32[5], hash32[6], hash32[7]);
		}

		free(temp_hashes);

		printf("Debug done\n");
		exit(1);
		*/



		*hashes_done = pdata[19] - first_nonce + throughput;

		if (work->nonces[0] != 0)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t _ALIGN(64) vhash[8];
			be32enc(&endiandata[19], work->nonces[0]);

			nightcap_hash(vhash, endiandata, work->nonces[0], work->height);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					nightcap_hash(vhash, endiandata, work->nonces[1], work->height);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}

				#ifdef CLOVERHASH_COMPARE_ON_CPU
				free(temp_hashes);
				#endif
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU (%u > %u)!", work->nonces[0], vhash[7], Htarg);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart && !abort_flag);

	#ifdef CLOVERHASH_COMPARE_ON_CPU
	free(temp_hashes);
	#endif

	*hashes_done = pdata[19] - first_nonce;

	return 0;
}
